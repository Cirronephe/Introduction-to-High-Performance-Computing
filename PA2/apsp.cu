#include "hip/hip_runtime.h"
// PLEASE MODIFY THIS FILE TO IMPLEMENT YOUR SOLUTION

// Brute Force APSP Implementation:

#include "apsp.h"
#include <cstdio>

template <typename T>
inline T ceiling(T x, T y) {
    return (x + y - 1) / y;
}

namespace {

__global__ void kernel_phase1(int p, int b, int n, int *graph) {
    const int TD = 1;
    int ty = threadIdx.y * TD;
    int tx = threadIdx.x * TD;
    int i = p * b + ty;
    int j = p * b + tx;

    __shared__ int shared_block[32][32];

    for (int u = 0; u < TD; ++u) {
        int pi = i + u;
        if (pi < n)
            for (int v = 0; v < TD; ++v) {
                int pj = j + v;
                if (pj < n)
                    shared_block[ty + u][tx + v] = graph[pi * n + pj];
            }
    }
    __syncthreads();

    int m = min(n - p * b, b);
    for (int k = 0; k < m; ++k) {
        for (int u = 0; u < TD; ++u)
            for (int v = 0; v < TD; ++v)
                shared_block[ty + u][tx + v] = min(shared_block[ty + u][tx + v], shared_block[ty + u][k] + shared_block[k][tx + v]);
        __syncthreads();
    }

    for (int u = 0; u < TD; ++u) {
        int pi = i + u;
        if (pi < n)    
            for (int v = 0; v < TD; ++v) {
                int pj = j + v;
                if (pj < n)
                    graph[pi * n + pj] = shared_block[ty + u][tx + v];
            }
    }
}

__global__ void kernel_phase2_row(int p, int b, int n, int *graph) {
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int i = p * b + ty;
    int j = blockIdx.x * blockDim.x + tx;
    
    if (j >= p * b) j += b;

    __shared__ int shared_block[32][32], shared_pivot[32][32];

    int pi = p * b + ty;
    int pj = p * b + tx;

    if (pi < n && pj < n) {
        shared_pivot[ty][tx] = graph[pi * n + pj];
    }
    if (i < n && j < n) {
        shared_block[ty][tx] = graph[i * n + j];
    }
    __syncthreads();

    int m = min(n - p * b, b);
    for (int k = 0; k < m; ++k) {
        shared_block[ty][tx] = min(shared_block[ty][tx], shared_pivot[ty][k] + shared_block[k][tx]);
        __syncthreads();
    }
    
    if (i < n && j < n) {
        graph[i * n + j] = shared_block[ty][tx];
    }
}

__global__ void kernel_phase2_col(int p, int b, int n, int *graph) {
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int i = blockIdx.y * blockDim.y + ty;
    int j = p * b + tx;
    
    if (i >= p * b) i += b;

    __shared__ int shared_block[32][32], shared_pivot[32][32];

    int pi = p * b + ty;
    int pj = p * b + tx;

    if (pi < n && pj < n) {
        shared_pivot[ty][tx] = graph[pi * n + pj];
    }
    if (i < n && j < n) {
        shared_block[ty][tx] = graph[i * n + j];
    }
    __syncthreads();

    int m = min(n - p * b, b);
    for (int k = 0; k < m; ++k) {
        shared_block[ty][tx] = min(shared_block[ty][tx], shared_block[ty][k] + shared_pivot[k][tx]);
        __syncthreads();
    }
    
    if (i < n && j < n) {
        graph[i * n + j] = shared_block[ty][tx];
    }
}

__global__ void kernel_phase3(int p, int b, int n, int *graph) {
    const int TD = 2;
    int ty = threadIdx.y * TD;
    int tx = threadIdx.x * TD;
    int i = blockIdx.y * blockDim.y * TD + ty;
    int j = blockIdx.x * blockDim.x * TD + tx;
    
    if (i >= p * b) i += b;
    if (j >= p * b) j += b;

    __shared__ int shared_block[32][32], shared_pivot_row[32][32], shared_pivot_col[32][32];
    
    #pragma unroll
    for (int u = 0; u < TD; ++u) {
        int pi = i + u;
        if (pi < n)
            #pragma unroll
            for (int v = 0; v < TD; ++v) {
                int pj = p * b + tx + v;
                if (pj < n)
                    shared_pivot_row[ty + u][tx + v] = graph[pi * n + pj];
            }
    }
    #pragma unroll
    for (int u = 0; u < TD; ++u) {
        int pi = p * b + ty + u;
        if (pi < n)
            #pragma unroll
            for (int v = 0; v < TD; ++v) {
                int pj = j + v;
                if (pj < n)
                    shared_pivot_col[ty + u][tx + v] = graph[pi * n + pj];
            }
    }
    #pragma unroll
    for (int u = 0; u < TD; ++u) {
        int pi = i + u;
        if (pi < n)
            #pragma unroll
            for (int v = 0; v < TD; ++v) {
                int pj = j + v;
                if (pj < n)
                    shared_block[ty + u][tx + v] = graph[pi * n + pj];
            }
    }
    __syncthreads();

    int reg_block[TD][TD];

    int m = min(n - p * b, b);
    #pragma unroll
    for (int u = 0; u < TD; ++u)
        #pragma unroll
        for (int v = 0; v < TD; ++v)
            reg_block[u][v] = shared_block[ty + u][tx + v];

    #pragma unroll
    for (int k = 0; k < m; ++k) {
        #pragma unroll
        for (int u = 0; u < TD; ++u)
            #pragma unroll
            for (int v = 0; v < TD; ++v)
                reg_block[u][v] = min(reg_block[u][v], shared_pivot_row[ty + u][k] + shared_pivot_col[k][tx + v]);
    }
    
    #pragma unroll
    for (int u = 0; u < TD; ++u) {
        int pi = i + u;
        if (pi < n)
            #pragma unroll
            for (int v = 0; v < TD; ++v) {
                int pj = j + v;
                if (pj < n)
                    graph[pi * n + pj] = reg_block[u][v];
            }
    }
}

}

void apsp(int n, /* device */ int *graph) {
    int b = 32, m = ceiling(n, b);
    for (int p = 0; p < m; ++p) {
        dim3 thr(32, 32);
        kernel_phase1<<<dim3(1, 1), dim3(32, 32)>>>(p, b, n, graph);
        kernel_phase2_row<<<dim3(m - 1, 1), thr>>>(p, b, n, graph);
        kernel_phase2_col<<<dim3(1, m - 1), thr>>>(p, b, n, graph);
        kernel_phase3<<<dim3(m - 1, m - 1), dim3(16, 16)>>>(p, b, n, graph);
    }
}

